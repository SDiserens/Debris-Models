#include "hip/hip_runtime.h"
// HootsFilter_GPU.cu : contains the implementation of the GPU elements of the Hoots collision algorithm.
//

#include "stdafx.h"
#include "HootsFilter.h"

// CUDA standard includes
#include <windows.h>
#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>
#include "math.h"
#include ""
#include <thrust\for_each.h>
#include <thrust\execution_policy.h>
#include <thrust\device_vector.h>
#include <thrust\remove.h>


struct PairKernel {
	int n;
	DebrisObject * population;
	PairKernel(int numObjects, DebrisObject *_data) {
		n = numObjects;
		population = _data;
	}
	template <typename Tuple>
	__device__ void operator()(Tuple t) {
		int x, y, z, i;
		i = thrust::get<0>(t);
		z = n - 1;
		x = 1;
		while (i > z) {
			i -= z;
			--z;
			++x;
		}
		y = x + i;

		thrust::get<1>(t).SetCollisionPair(population[--x], population[--y]);
	}
};

struct PAKernel {
	double pAThreshold;
	PAKernel(double threshold) {
		pAThreshold = threshold;
	}
	__device__ bool operator()(CollisionPair objectPair) {
		double maxPerigee, minApogee;
		// Perigee Apogee Test
		maxPerigee = max(objectPair.primaryElements.GetPerigee(), objectPair.secondaryElements.GetPerigee());
		minApogee = min(objectPair.primaryElements.GetApogee(), objectPair.secondaryElements.GetApogee());

		return (maxPerigee - minApogee) <= max(pAThreshold, objectPair.GetBoundingRadii());
	}
};

__host__ thrust::device_vector<CollisionPair> HootsFilter::CreatePairList_GPU(DebrisPopulation & population)
{
	int n = population.GetPopulationSize();
	int N = n*(n - 1) / 2;
	thrust::device_vector<CollisionPair> pairList(N);
	thrust::device_vector<DebrisObject> populationList;

	for_each(population.population.begin(), population.population.end(), [&](pair<long, DebrisObject> object) {
		populationList.push_back(object.second);
	});

	thrust::counting_iterator<int> first(1);
	thrust::counting_iterator<int> last = first + N;

	thrust::for_each(thrust::make_zip_iterator(thrust::make_tuple(first, pairList.begin())), thrust::make_zip_iterator(thrust::make_tuple(last, pairList.end())), PairKernel(n, thrust::raw_pointer_cast(populationList.data())));

	int p = thrust::copy_if(thrust::device, pairList.begin(), pairList.end(), pairList.begin(), PAKernel(pAThreshold)) - pairList.begin();
	pairList.resize(p);

	return pairList;
}


struct Collision
{
	__host__ __device__
		bool operator()(CollisionPair objectPair)
	{
		return (objectPair.collision);
	}
};

struct GeometricFilterKernel {
	double threshold;
	GeometricFilterKernel(double threshold) {
		threshold = threshold;
	}
	__device__ bool operator()(CollisionPair &objectPair) {
		objectPair.collision = (objectPair.minSeperation < threshold);

		return objectPair.collision;
	}
};

struct TimeFilterKernel {
	double timeStep;
	TimeFilterKernel(double timestep) {
		timeStep = timestep;
	}
	__device__ bool operator()(CollisionPair &objectPair) {
		objectPair.collision = false;


		return objectPair.collision;
	}
};

void HootsFilter::MainCollision_GPU(DebrisPopulation & population, double timestep)
{
	double mass, tempProbability, epoch = population.GetEpoch();
	Event tempEvent;
	pair<long, long> pairID;
	vector<double> candidateTimeList, collisionTimes;
	double closeTime, closeApproach;
	bool collide;

	// Filter Cube List
	thrust::device_vector<CollisionPair> pairListIn = CreatePairList_GPU(population);
	timeStep = timestep;
	//unsigned int numThreads, numBlocks;
	//computeGridSize(pairList.size(), 256, numBlocks, numThreads);
	size_t n = pairListIn.size();
	thrust::device_vector<CollisionPair> pairList(n);

	thrust::host_vector<CollisionPair> outList(pairList.begin(), pairList.end());
	concurrency::parallel_for_each(outList.begin(), outList.end(), [&](CollisionPair& objectPair)
	{	switch (MOIDtype) {
	case 0:
		objectPair.minSeperation = objectPair.CalculateMinimumSeparation();
		break;
	case 1:
		objectPair.minSeperation = objectPair.CalculateMinimumSeparation_DL();
		break;
	case 2:
		objectPair.minSeperation = objectPair.CalculateMinimumSeparation_MOID();
		break;
	}
	});

	pairList = thrust::device_vector<CollisionPair>(outList.begin(), outList.end());

	thrust::for_each(thrust::device, pairListIn.begin(), pairListIn.end(), GeometricFilterKernel(timestep));
	n = thrust::copy_if(thrust::device, pairListIn.begin(), pairListIn.end(), pairList.begin(), Collision()) - pairList.begin();
	pairList.resize(n);
	outList = thrust::host_vector<CollisionPair>(pairList.begin(), pairList.end());

	for (int i = 0; i < outList.size(); i++) {
		CollisionPair objectPair = outList[i];
		candidateTimeList.clear();
		collisionTimes.clear();

		//TODO - ADD GPU code for time/coplanar filter
		if (objectPair.GetRelativeInclination() == 0)
			candidateTimeList.push_back(-1);
		else
		{
			candidateTimeList = TimeFilter(objectPair, timeStep);
		}

		if (candidateTimeList.size() > 0)
		{
			if (candidateTimeList[0] < 0)
				candidateTimeList = CoplanarFilter(objectPair, timeStep);
		}

		if (candidateTimeList.size() > 0)
		{
			//vector<double> altitudes;
			//collisionTimes = DetermineCollisionTimes(objectPair, candidateTimeList, altitudes);

			pairID = make_pair(objectPair.primaryID, objectPair.secondaryID);
			mass = objectPair.primaryMass + objectPair.secondaryMass;
			for (double candidateTime : candidateTimeList)
			{
				closeTime = CalculateClosestApproachTime(objectPair, candidateTime);
				closeApproach = objectPair.CalculateSeparationAtTime(closeTime);
				collide = closeApproach < (objectPair.GetBoundingRadii() + collisionThreshold);
				if (outputTimes) {

					Event tempEvent(population.GetEpoch() + closeTime, pairID.first, pairID.second, objectPair.GetRelativeVelocity(), mass, objectPair.GetCollisionAltitude(), closeApproach);
					newCollisionTimes.push_back(collide);
					newCollisionList.push_back(tempEvent);
				}
				else if (collide)
				{
					Event tempEvent(population.GetEpoch() + closeTime, pairID.first, pairID.second, objectPair.GetRelativeVelocity(), mass, objectPair.GetCollisionAltitude(), closeApproach);
					newCollisionList.push_back(tempEvent); // Note in this scenario only adds once regardless of number of # potential collisions for pair
					break;
				}
			}
		}
	}
}