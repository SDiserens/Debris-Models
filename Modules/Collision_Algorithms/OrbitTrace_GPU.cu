#include "hip/hip_runtime.h"
// OrbitTrace.cpp : contains the implementation of the GPU elements of the Orbit Trace collision algorithm.
//

#include "stdafx.h"
#include "OrbitTrace.h"

// CUDA standard includes
#include <windows.h>
#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>
#include "math.h"
#include ""
#include <thrust\for_each.h>
#include <thrust\execution_policy.h>
#include <thrust\device_vector.h>
#include <thrust\remove.h>

#define CUDASTRIDE 256
typedef thrust::device_vector<CollisionPair>::iterator dvit;

//Round a / b to nearest higher integer value
unsigned int  iDivUp(unsigned int  a, unsigned int  b)
{
	return (a % b != 0) ? (a / b + 1) : (a / b);
}

// compute grid and thread block size for a given number of elements
void computeGridSize(unsigned int n, unsigned int blockSize, unsigned int  &numBlocks, unsigned int  &numThreads)
{
	numThreads = min(blockSize, n);
	numBlocks = iDivUp(n, numThreads);
}


struct PairKernel {
	int n;
	DebrisObject * population;
	PairKernel(int numObjects, DebrisObject *_data) {
		n = numObjects;
		population = _data;
	}
	template <typename Tuple>
	__device__ void operator()(Tuple t) {
		int x, y, z, i;
		i = thrust::get<0>(t);
		z = n - 1;
		x = 1;
		while (i > z) {
			i -= z;
			--z;
			++x;
		}
		y = x + i;

		thrust::get<1>(t).SetCollisionPair(population[--x], population[--y]);
	}
};

struct PAKernel {
	double pAThreshold;
	PAKernel(double threshold) {
		pAThreshold = threshold;
	}
	__device__ bool operator()(CollisionPair objectPair) {
		double maxPerigee, minApogee;
		// Perigee Apogee Test
		maxPerigee = max(objectPair.primaryElements.GetPerigee(), objectPair.secondaryElements.GetPerigee());
		minApogee = min(objectPair.primaryElements.GetApogee(), objectPair.secondaryElements.GetApogee());

		return (maxPerigee - minApogee) <= max(pAThreshold, objectPair.GetBoundingRadii());
	}
};

__host__ thrust::device_vector<CollisionPair> OrbitTrace::CreatePairList_GPU(DebrisPopulation & population)
{
	int n = population.GetPopulationSize();
	int N = n*(n - 1) / 2;
	thrust::device_vector<CollisionPair> pairList(N);
	thrust::device_vector<DebrisObject> populationList;
	
	for_each(population.population.begin(), population.population.end(), [&](pair<long, DebrisObject> object) {
		populationList.push_back(object.second);
	});

	thrust::counting_iterator<int> first(1);
	thrust::counting_iterator<int> last = first + N;

	thrust::for_each(thrust::make_zip_iterator(thrust::make_tuple(first, pairList.begin())), thrust::make_zip_iterator(thrust::make_tuple(last, pairList.end())), PairKernel(n, thrust::raw_pointer_cast(populationList.data())));

	int p = thrust::copy_if(thrust::device, pairList.begin(), pairList.end(), pairList.begin(), PAKernel(pAThreshold)) - pairList.begin();

	populationList.clear();
	populationList.shrink_to_fit();

	pairList.resize(p);
	pairList.shrink_to_fit();

	return pairList;
}

__host__ thrust::device_vector<CollisionPair> OrbitTrace::CreatePairList_CPU(DebrisPopulation & population)
{

	thrust::device_vector<CollisionPair> pairList;
	mutex mtx;
	concurrency::parallel_for_each(population.population.begin(), population.population.end(), [&](auto& it) {
		auto jt = population.population.find(it.first);
		for (++jt; jt != population.population.end(); ++jt)
		{
			/// Add pair to list
			//DebrisObject& primaryObject(population.Ge), secondaryObject;
			CollisionPair pair(it.second, jt->second);
			if (PerigeeApogeeTest(pair)) {
				mtx.lock();
				pairList.push_back(pair);
				mtx.unlock();
			}
			else
				pair.~CollisionPair();
		}
	});
	return pairList;
}

/*
__host__ void OrbitTrace::MainCollision_GPU_Cuda(DebrisPopulation & population, double timestep)
{
	double tempProbability, collisionRate, altitude, mass;
	list<CollisionPair> pairList;
	pair<long, long> pairID;
	bool collision;

	// Filter Cube List
	pairList = CreatePairList_GPU(population);
	timeStep = timestep;
	unsigned int numThreads, numBlocks;
	computeGridSize(pairList.size(), 256, numBlocks, numThreads);
	//- Add code for GPU use
		// 1D iteration over pairList
		//int index = blockIdx.x * blockDim.x + threadIdx.x;

}
*/

__device__ bool HeadOnFilter(CollisionPair objectPair)
{
	bool headOn = false;
	double deltaW;
	double eLimitP = objectPair.GetBoundingRadii() / objectPair.primaryElements.semiMajorAxis;
	double eLimitS = objectPair.GetBoundingRadii() / objectPair.secondaryElements.semiMajorAxis;
	// OT Head on filter
	if ((objectPair.primaryElements.eccentricity <= eLimitP) && (objectPair.secondaryElements.eccentricity <= eLimitS))
		headOn = true;
	else
	{
		deltaW = fabs((double)(Pi - objectPair.primaryElements.argPerigee - objectPair.secondaryElements.argPerigee));
		if (deltaW <= 1)
			headOn = true;
		else if (Tau - deltaW <= 1)
			headOn = true;
	}

	return headOn;
}

__device__ bool SynchronizedFilter(CollisionPair objectPair, double timeStep)
{
	double meanMotionP, meanMotionS, driftAngle;
	// OT synch filter
	meanMotionP = Tau / objectPair.primaryElements.CalculatePeriod();
	meanMotionS = Tau / objectPair.secondaryElements.CalculatePeriod();

	driftAngle = fabs(meanMotionP - meanMotionS) * timeStep;
	return (driftAngle >= Tau);
}

__device__ bool ProximityFilter(CollisionPair objectPair)
{
	//  OT  proximity filter
	double deltaMP, deltaMS, deltaMAngle, deltaMLinear, combinedSemiMajorAxis;
	OrbitalAnomalies anomaliesP, anomaliesS;

	anomaliesP.SetTrueAnomaly(objectPair.approachAnomalyP);
	anomaliesS.SetTrueAnomaly(objectPair.approachAnomalyS);

	deltaMP = fabs(anomaliesP.GetMeanAnomaly(objectPair.primaryElements.eccentricity) - objectPair.primaryElements.GetMeanAnomaly());
	deltaMS = fabs(anomaliesS.GetMeanAnomaly(objectPair.secondaryElements.eccentricity) - objectPair.secondaryElements.GetMeanAnomaly());

	combinedSemiMajorAxis = (objectPair.primaryElements.semiMajorAxis + objectPair.secondaryElements.semiMajorAxis) / 2;
	deltaMAngle = fabs(deltaMP - deltaMS);
	deltaMLinear = deltaMAngle * combinedSemiMajorAxis;

	return (deltaMLinear <= objectPair.GetBoundingRadii());
}

struct Collision
{
	__host__ __device__
		bool operator()(CollisionPair objectPair)
	{
		return (!objectPair.collision);
	}
};

struct CollisionFilterKernel {
	double timeStep;
	CollisionFilterKernel(double timestep) {
		timeStep = timestep;
	}
	__device__ void operator()(CollisionPair &objectPair) {
		objectPair.collision = false;

		double combinedSemiMajorAxis = objectPair.primaryElements.semiMajorAxis + objectPair.secondaryElements.semiMajorAxis;
		bool coplanar = objectPair.relativeInclination <= (2 * asin(objectPair.boundingRadii / combinedSemiMajorAxis));
		objectPair.coplanar = coplanar;

		if (coplanar)
		{
			// Calculate orbit intersections for coplanar
			objectPair.CalculateArgumenstOfIntersectionCoplanar();
			if (HeadOnFilter(objectPair) || !SynchronizedFilter(objectPair, timeStep) || ProximityFilter(objectPair))
				objectPair.collision = true;
		}
		else
		{
			// Calculate intersections for non coplanar
			objectPair.CalculateArgumenstOfIntersection();
			if (!SynchronizedFilter(objectPair, timeStep) || ProximityFilter(objectPair))
				objectPair.collision = true;
			if (objectPair.collision) {
			}
		}
	}
};
struct LowerBoundFilter {
	double pAThreshold;
	LowerBoundFilter(double threshold) {
		pAThreshold = threshold;
	}
	__device__ void operator()(CollisionPair &objectPair) {
		if (objectPair.CalculateLowerBoundSeparation() > max(pAThreshold, objectPair.boundingRadii))
		{
			objectPair.collision = false;
		}
	}
};
struct MinSeperation {
	MinSeperation() {};
	__device__ void operator()(CollisionPair &objectPair) {
		double sep = objectPair.CalculateMinimumSeparation();

		//return objectPair.minSeperation;
	}
};

struct CollisionRateKernel {
		double timeStep, pAThreshold;
		bool relativeGravity;
		CollisionRateKernel(double timestep, double threshold, bool relG) {
			timeStep = timestep;
			pAThreshold = threshold;
			relativeGravity = relG;
		}
		__device__ double operator()(CollisionPair &objectPair) {

			double collisionRate, threshold, boundingRadii, relativeVelocity, scaling, escapeVelocity2, gravitationalPerturbation, collisionRate2;
			
			boundingRadii = objectPair.GetBoundingRadii();
			vector3D velocityI, velocityJ;
			boundingRadii = objectPair.GetBoundingRadii();
			threshold = max(pAThreshold, boundingRadii);
			//sinAngle = velocityI.VectorCrossProduct(velocityJ).vectorNorm() / (velocityI.vectorNorm() * velocityJ.vectorNorm());
			scaling = 1;
			if (boundingRadii < pAThreshold) {
				scaling = boundingRadii / pAThreshold;
				scaling = scaling * scaling;
			}
			if (relativeGravity)
			{
				escapeVelocity2 = 2 * (objectPair.primaryMass + objectPair.secondaryMass) * GravitationalConstant / boundingRadii;
			}

			// OT collision rate
			if (objectPair.minSeperation < threshold)
			{
				objectPair.primaryElements.SetTrueAnomaly(objectPair.approachAnomalyP);
				objectPair.secondaryElements.SetTrueAnomaly(objectPair.approachAnomalyS);
				velocityI = objectPair.primaryElements.GetVelocity();
				velocityJ = objectPair.secondaryElements.GetVelocity();
				relativeVelocity = objectPair.relativeVelocity = velocityI.CalculateRelativeVector(velocityJ).vectorNorm();
				if (relativeGravity)
				{
					gravitationalPerturbation = (1 + escapeVelocity2 / (relativeVelocity * relativeVelocity));
				}
				else
					gravitationalPerturbation = 1;

				collisionRate = gravitationalPerturbation * Pi * threshold * relativeVelocity /
					(2 * velocityI.VectorCrossProduct(velocityJ).vectorNorm()  * objectPair.primaryElements.CalculatePeriod() * objectPair.secondaryElements.CalculatePeriod());
			}
			else
				collisionRate = 0;

			if (objectPair.minSeperation2 < threshold)
			{

				objectPair.primaryElements.SetTrueAnomaly(objectPair.approachAnomalyP2);
				objectPair.secondaryElements.SetTrueAnomaly(objectPair.approachAnomalyS2);
				velocityI = objectPair.primaryElements.GetVelocity();
				velocityJ = objectPair.secondaryElements.GetVelocity();
				relativeVelocity = objectPair.relativeVelocity2 = velocityI.CalculateRelativeVector(velocityJ).vectorNorm();
				if (relativeGravity)
				{
					gravitationalPerturbation = (1 + escapeVelocity2 / (relativeVelocity * relativeVelocity));
				}
				else
					gravitationalPerturbation = 1;

				collisionRate2 = gravitationalPerturbation * Pi * threshold * relativeVelocity /
					(2 * velocityI.VectorCrossProduct(velocityJ).vectorNorm()  * objectPair.primaryElements.CalculatePeriod() * objectPair.secondaryElements.CalculatePeriod());
			}
			else
				collisionRate2 = 0;
 
			objectPair.probability = timeStep * scaling * (collisionRate + collisionRate2);
			return objectPair.probability;
		}
};

__host__ void OrbitTrace::MainCollision_GPU(DebrisPopulation & population, double timestep)
{
	double mass, tempProbability, epoch = population.GetEpoch(), sep;
	Event tempEvent;

	// Filter Cube List
	thrust::device_vector<CollisionPair> pairList = CreatePairList_GPU(population);
	timeStep = timestep;
	//unsigned int numThreads, numBlocks;
	//computeGridSize(pairList.size(), 256, numBlocks, numThreads);
	size_t n = pairList.size();
	thrust::for_each(thrust::device, pairList.begin(), pairList.end(), CollisionFilterKernel(timestep));
	
	n = thrust::remove_if(pairList.begin(), pairList.end(), Collision()) - pairList.begin();
	pairList.resize(n);

	thrust::for_each(thrust::device, pairList.begin(), pairList.end(), LowerBoundFilter(pAThreshold));
	n = thrust::remove_if(pairList.begin(), pairList.end(), Collision()) - pairList.begin();
	pairList.resize(n);

	thrust::host_vector<CollisionPair> outList(pairList.begin(), pairList.end());
	pairList.clear();
	pairList.shrink_to_fit();
	//n = std::remove_if(outList.begin(), outList.end(), Collision()) - outList.begin();
	//outList.resize(n);

	concurrency::parallel_for_each(outList.begin(), outList.end(), [&](CollisionPair& objectPair)
	{	
		switch (MOIDtype) {
		case 0:
			sep = objectPair.CalculateMinimumSeparation();
			break;
		case 1:
			sep = objectPair.CalculateMinimumSeparation_DL(max_root_error, min_root_error, max_anom_error);
			break;
		case 2:
			sep = objectPair.CalculateMinimumSeparation_MOID();
			break;
		}
	});

	pairList = thrust::device_vector<CollisionPair>(outList.begin(), outList.end());
	
	//thrust::for_each(thrust::device, pairList.begin(), pairList.end(), MinSeperation());

	thrust::device_vector<double> probabilityList(n);
	thrust::transform(thrust::device, pairList.begin(), pairList.end(), probabilityList.begin(), CollisionRateKernel(timestep, pAThreshold, relativeGravity));

	//dvit collisionEnd = thrust::remove_if(thrust::device, pairList.begin(), pairList.end(), NotCollision());

	outList = thrust::host_vector<CollisionPair>(pairList.begin(), pairList.end());
	//thrust::host_vector<CollisionPair> outList(pairList.begin(), pairList.end());
	thrust::host_vector<double> pList(probabilityList.begin(), probabilityList.end());
	pairList.clear();
	pairList.shrink_to_fit();
	probabilityList.clear();
	probabilityList.shrink_to_fit();
	//thrust::host_vector<double> probOut(probabilityList.begin(), probabilityList.end());

	for (int i = 0; i < outList.size(); i++) {
		CollisionPair objectPair = outList[i];
		tempProbability = pList[i];
		if (tempProbability > 0) {
			mass = objectPair.primaryMass + objectPair.secondaryMass;
			tempEvent = Event(epoch, objectPair.primaryID, objectPair.secondaryID, objectPair.GetRelativeVelocity(), mass, objectPair.GetCollisionAltitude(), objectPair.GetMinSeparation(), tempProbability);
			tempEvent.SetCollisionAnomalies(objectPair.approachAnomalyP, objectPair.approachAnomalyS);
			// Store Collisions 
			newCollisionList.push_back(tempEvent);
		}
	}
	outList.clear();
	outList.shrink_to_fit();
	pList.clear();
	pList.shrink_to_fit();

	elapsedTime += timeStep;

}

