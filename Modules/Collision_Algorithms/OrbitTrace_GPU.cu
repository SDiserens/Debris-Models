#include "hip/hip_runtime.h"
// OrbitTrace.cpp : contains the implementation of the GPU elements of the Orbit Trace collision algorithm.
//

#include "stdafx.h"
#include "OrbitTrace.h"

// CUDA standard includes
#include <windows.h>
#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>
#include "math.h"
#include ""
#include <thrust\for_each.h>
#include <thrust\execution_policy.h>
#include <thrust\device_vector.h>

#define CUDASTRIDE 256

//Round a / b to nearest higher integer value
unsigned int  iDivUp(unsigned int  a, unsigned int  b)
{
	return (a % b != 0) ? (a / b + 1) : (a / b);
}

// compute grid and thread block size for a given number of elements
void computeGridSize(unsigned int n, unsigned int blockSize, unsigned int  &numBlocks, unsigned int  &numThreads)
{
	numThreads = min(blockSize, n);
	numBlocks = iDivUp(n, numThreads);
}

__device__ double CollisionRate(CollisionPair &objectPair, double pAThreshold)
{
	double collisionRate, boundingRadii, minSeperation, relativeVelocity;
	vector3D velocityI, velocityJ;

	minSeperation = objectPair.CalculateMinimumSeparation();

	velocityI = objectPair.primary.GetVelocity();
	velocityJ = objectPair.secondary.GetVelocity();

	relativeVelocity = velocityI.CalculateRelativeVector(velocityJ).vectorNorm();
	boundingRadii = max(pAThreshold, objectPair.GetBoundingRadii());
	objectPair.SetRelativeVelocity(relativeVelocity);
	//sinAngle = velocityI.VectorCrossProduct(velocityJ).vectorNorm() / (velocityI.vectorNorm() * velocityJ.vectorNorm());

	// OT collision rate
	if (boundingRadii > minSeperation)
		collisionRate = Pi * boundingRadii * relativeVelocity /
		(2 * velocityI.VectorCrossProduct(velocityJ).vectorNorm()  * objectPair.primary.GetPeriod() * objectPair.secondary.GetPeriod());
	else
		collisionRate = 0;

	return collisionRate;
}
__host__ thrust::device_vector<CollisionPair> OrbitTrace::CreatePairList_GPU(DebrisPopulation & population)
{
	thrust::device_vector<CollisionPair> pairList;
	// TODO - GPU code for creating pairList
	//Talk to Pete about i, j where i < j < N
	for (auto it = population.population.begin(); it != population.population.end(); it++)
	{
		// For each subsequent object
		auto jt = it;
		for (++jt; jt != population.population.end(); ++jt)
		{
			/// Add pair to list
			//DebrisObject& primaryObject(population.Ge), secondaryObject;
			CollisionPair pair(it->second, jt->second);
			if (PerigeeApogeeTest(pair))
				pairList.push_back(pair);
			else
				pair.~CollisionPair();
		}
	}

	return pairList;
}

/*
//TODO - add device function  to operate on each collision pair
__device__ void OrbitTraceAlgorithm(list<CollisionPair>& pairList) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	
}

__host__ void OrbitTrace::MainCollision_GPU_Cuda(DebrisPopulation & population, double timestep)
{
	double tempProbability, collisionRate, altitude, mass;
	list<CollisionPair> pairList;
	pair<long, long> pairID;
	bool collision;

	// Filter Cube List
	pairList = CreatePairList_GPU(population);
	timeStep = timestep;
	unsigned int numThreads, numBlocks;
	computeGridSize(pairList.size(), 256, numBlocks, numThreads);
	//TODO - Add code for GPU use
		// 1D iteration over pairList
		//int index = blockIdx.x * blockDim.x + threadIdx.x;

}
*/
__device__ bool HeadOnFilter(CollisionPair& objectPair)
{
	bool headOn = false;
	double deltaW;
	double eLimitP = objectPair.GetBoundingRadii() / objectPair.primary.GetElements().semiMajorAxis;
	double eLimitS = objectPair.GetBoundingRadii() / objectPair.secondary.GetElements().semiMajorAxis;
	// OT Head on filter
	if ((objectPair.primary.GetElements().eccentricity <= eLimitP) && (objectPair.secondary.GetElements().eccentricity <= eLimitS))
		headOn = true;
	else
	{
		deltaW = fabs((double)(Pi - objectPair.primary.GetElements().argPerigee - objectPair.secondary.GetElements().argPerigee));
		if (deltaW <= 1)
			headOn = true;
		else if (Tau - deltaW <= 1)
			headOn = true;
	}

	return headOn;
}

__device__ bool SynchronizedFilter(CollisionPair& objectPair, double timeStep)
{
	double meanMotionP, meanMotionS, driftAngle;
	// OT synch filter
	meanMotionP = Tau / objectPair.primary.GetPeriod();
	meanMotionS = Tau / objectPair.secondary.GetPeriod();

	driftAngle = fabs(meanMotionP - meanMotionS) * timeStep;
	return (driftAngle >= Tau);
}

__device__ bool ProximityFilter(CollisionPair& objectPair)
{
	//  OT  proximity filter
	double deltaMP, deltaMS, deltaMAngle, deltaMLinear, combinedSemiMajorAxis;
	OrbitalAnomalies anomaliesP, anomaliesS;

	anomaliesP.SetTrueAnomaly(objectPair.approachAnomalyP);
	anomaliesS.SetTrueAnomaly(objectPair.approachAnomalyS);

	deltaMP = fabs(anomaliesP.GetMeanAnomaly(objectPair.primary.GetElements().eccentricity) - objectPair.primary.GetElements().GetMeanAnomaly());
	deltaMS = fabs(anomaliesS.GetMeanAnomaly(objectPair.secondary.GetElements().eccentricity) - objectPair.secondary.GetElements().GetMeanAnomaly());

	combinedSemiMajorAxis = (objectPair.primary.GetElements().semiMajorAxis + objectPair.secondary.GetElements().semiMajorAxis) / 2;
	deltaMAngle = fabs(deltaMP - deltaMS);
	deltaMLinear = deltaMAngle * combinedSemiMajorAxis;

	return (deltaMLinear <= objectPair.GetBoundingRadii());
}


struct CollisionFilterKernel {
	double timeStep;
	CollisionFilterKernel(double timestep) {
		timeStep = timestep;
	}
	__device__ void operator()(CollisionPair& objectPair) {
		objectPair.collision = false;

		objectPair.CalculateRelativeInclination();
		double combinedSemiMajorAxis = objectPair.primary.GetElements().semiMajorAxis + objectPair.secondary.GetElements().semiMajorAxis;
		bool coplanar = objectPair.GetRelativeInclination() <= (2 * asin(objectPair.GetBoundingRadii() / combinedSemiMajorAxis));
		objectPair.coplanar = coplanar;

		if (coplanar)
		{
			// Calculate orbit intersections for coplanar
			objectPair.CalculateArgumenstOfIntersectionCoplanar();
			if (HeadOnFilter(objectPair) || !SynchronizedFilter(objectPair, timeStep) || ProximityFilter(objectPair))
				objectPair.collision = true;
		}
		else
		{
			// Calculate intersections for non coplanar
			objectPair.CalculateArgumenstOfIntersection();
			if (!SynchronizedFilter(objectPair, timeStep) || ProximityFilter(objectPair))
				objectPair.collision = true;
		}
	}
};

struct CollisionRateKernel {
		double timeStep, pAThreshold;
		CollisionRateKernel(double timestep, double threshold) {
			timeStep = timestep;
			pAThreshold = threshold;
		}
	__device__ void operator()(CollisionPair& objectPair) {
		if (objectPair.collision)
		{
			double  altitude, mass;
			thrust::pair<long, long> pairID;
			objectPair.probability = timeStep * CollisionRate(objectPair, pAThreshold);
			pairID = thrust::make_pair(objectPair.primaryID, objectPair.secondaryID);

			altitude = objectPair.primary.GetElements().GetRadialPosition();
			mass = objectPair.primary.GetMass() + objectPair.secondary.GetMass();
			objectPair.tempEvent = Event(0, pairID.first, pairID.second, objectPair.GetRelativeVelocity(), mass, altitude);
		}
		else
			objectPair.probability = 0;
	}
};

__host__ void OrbitTrace::MainCollision_GPU(DebrisPopulation & population, double timestep)
{
	double tempProbability, epoch = population.GetEpoch();
	thrust::device_vector<CollisionPair> pairList;
	int i, j, n;

	// Filter Cube List
	pairList = CreatePairList_GPU(population);
	timeStep = timestep;
	//unsigned int numThreads, numBlocks;
	//computeGridSize(pairList.size(), 256, numBlocks, numThreads);
	//TODO - Add code for GPU use
	n = pairList.size();
	for (i = 0; i < n; i += CUDASTRIDE) {
		j = min(i + CUDASTRIDE, n);
		thrust::for_each(thrust::device, pairList.begin()+i, pairList.begin() + j, CollisionFilterKernel(timestep));
	}
	for (i = 0; i < n; i += CUDASTRIDE) {
		j = min(i + CUDASTRIDE, n);
		thrust::for_each(thrust::device, pairList.begin() + i, pairList.begin() + j, CollisionRateKernel(timestep, pAThreshold));
	}
	//thrust::host_vector<CollisionPair> pairList_local = pairList;

	for (int i = 0; i < pairList.size(); i++) {
		CollisionPair objectPair = pairList[i];
		tempProbability = objectPair.probability;
		//	-- Determine if collision occurs through MC (random number generation)
		if (outputProbabilities && tempProbability > 0)
		{
			//	-- Store collision probability
			objectPair.tempEvent.SetEpoch(epoch);
			newCollisionProbabilities.push_back(tempProbability);
			newCollisionList.push_back(objectPair.tempEvent);

		}
		else
		{
			if (DetermineCollision(tempProbability))
			{
				// Store Collisions 
				objectPair.tempEvent.SetEpoch(epoch);
				newCollisionList.push_back(objectPair.tempEvent);
			}
		}
	}

	elapsedTime += timeStep;

}

