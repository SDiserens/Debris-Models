#include "hip/hip_runtime.h"
// OrbitTrace.cpp : contains the implementation of the GPU elements of the Orbit Trace collision algorithm.
//

#include "stdafx.h"
#include "OrbitTrace.h"

// CUDA standard includes
#include <windows.h>
#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>
#include "math.h"
#include ""
#include <thrust\for_each.h>
#include <thrust\execution_policy.h>
#include <thrust\device_vector.h>
#include <thrust\remove.h>

#define CUDASTRIDE 256
typedef thrust::device_vector<CollisionPair>::iterator dvit;

//Round a / b to nearest higher integer value
unsigned int  iDivUp(unsigned int  a, unsigned int  b)
{
	return (a % b != 0) ? (a / b + 1) : (a / b);
}

// compute grid and thread block size for a given number of elements
void computeGridSize(unsigned int n, unsigned int blockSize, unsigned int  &numBlocks, unsigned int  &numThreads)
{
	numThreads = min(blockSize, n);
	numBlocks = iDivUp(n, numThreads);
}

__host__ thrust::device_vector<CollisionPair> OrbitTrace::CreatePairList_GPU(DebrisPopulation & population)
{
	thrust::device_vector<CollisionPair> pairList;
	// TODO - GPU code for creating pairList
	//Talk to Pete about i, j where i < j < N
	for (auto it = population.population.begin(); it != population.population.end(); it++)
	{
		// For each subsequent object
		auto jt = it;
		for (++jt; jt != population.population.end(); ++jt)
		{
			/// Add pair to list
			//DebrisObject& primaryObject(population.Ge), secondaryObject;
			CollisionPair pair(it->second, jt->second);
			if (PerigeeApogeeTest(pair))
				pairList.push_back(pair);
			else
				pair.~CollisionPair();
		}
	}

	return pairList;
}

/*
//TODO - add device function  to operate on each collision pair
__device__ void OrbitTraceAlgorithm(list<CollisionPair>& pairList) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	
}

__host__ void OrbitTrace::MainCollision_GPU_Cuda(DebrisPopulation & population, double timestep)
{
	double tempProbability, collisionRate, altitude, mass;
	list<CollisionPair> pairList;
	pair<long, long> pairID;
	bool collision;

	// Filter Cube List
	pairList = CreatePairList_GPU(population);
	timeStep = timestep;
	unsigned int numThreads, numBlocks;
	computeGridSize(pairList.size(), 256, numBlocks, numThreads);
	//TODO - Add code for GPU use
		// 1D iteration over pairList
		//int index = blockIdx.x * blockDim.x + threadIdx.x;

}
*/
__device__ bool HeadOnFilter(CollisionPair& objectPair)
{
	bool headOn = false;
	double deltaW;
	double eLimitP = objectPair.GetBoundingRadii() / objectPair.primary.GetElements().semiMajorAxis;
	double eLimitS = objectPair.GetBoundingRadii() / objectPair.secondary.GetElements().semiMajorAxis;
	// OT Head on filter
	if ((objectPair.primary.GetElements().eccentricity <= eLimitP) && (objectPair.secondary.GetElements().eccentricity <= eLimitS))
		headOn = true;
	else
	{
		deltaW = fabs((double)(Pi - objectPair.primary.GetElements().argPerigee - objectPair.secondary.GetElements().argPerigee));
		if (deltaW <= 1)
			headOn = true;
		else if (Tau - deltaW <= 1)
			headOn = true;
	}

	return headOn;
}

__device__ bool SynchronizedFilter(CollisionPair& objectPair, double timeStep)
{
	double meanMotionP, meanMotionS, driftAngle;
	// OT synch filter
	meanMotionP = Tau / objectPair.primary.GetPeriod();
	meanMotionS = Tau / objectPair.secondary.GetPeriod();

	driftAngle = fabs(meanMotionP - meanMotionS) * timeStep;
	return (driftAngle >= Tau);
}

__device__ bool ProximityFilter(CollisionPair& objectPair)
{
	//  OT  proximity filter
	double deltaMP, deltaMS, deltaMAngle, deltaMLinear, combinedSemiMajorAxis;
	OrbitalAnomalies anomaliesP, anomaliesS;

	anomaliesP.SetTrueAnomaly(objectPair.approachAnomalyP);
	anomaliesS.SetTrueAnomaly(objectPair.approachAnomalyS);

	deltaMP = fabs(anomaliesP.GetMeanAnomaly(objectPair.primary.GetElements().eccentricity) - objectPair.primary.GetElements().GetMeanAnomaly());
	deltaMS = fabs(anomaliesS.GetMeanAnomaly(objectPair.secondary.GetElements().eccentricity) - objectPair.secondary.GetElements().GetMeanAnomaly());

	combinedSemiMajorAxis = (objectPair.primary.GetElements().semiMajorAxis + objectPair.secondary.GetElements().semiMajorAxis) / 2;
	deltaMAngle = fabs(deltaMP - deltaMS);
	deltaMLinear = deltaMAngle * combinedSemiMajorAxis;

	return (deltaMLinear <= objectPair.GetBoundingRadii());
}

struct NotCollision
{
	__host__ __device__
		bool operator()(CollisionPair objectPair)
	{
		return (!objectPair.collision);
	}
};

struct CollisionFilterKernel {
	double timeStep;
	CollisionFilterKernel(double timestep) {
		timeStep = timestep;
	}
	__device__ bool operator()(CollisionPair& objectPair) {
		objectPair.collision = false;

		objectPair.CalculateRelativeInclination();
		double combinedSemiMajorAxis = objectPair.primary.GetElements().semiMajorAxis + objectPair.secondary.GetElements().semiMajorAxis;
		bool coplanar = objectPair.GetRelativeInclination() <= (2 * asin(objectPair.GetBoundingRadii() / combinedSemiMajorAxis));
		objectPair.coplanar = coplanar;

		if (coplanar)
		{
			// Calculate orbit intersections for coplanar
			objectPair.CalculateArgumenstOfIntersectionCoplanar();
			if (HeadOnFilter(objectPair) || !SynchronizedFilter(objectPair, timeStep) || ProximityFilter(objectPair))
				objectPair.collision = true;
		}
		else
		{
			// Calculate intersections for non coplanar
			objectPair.CalculateArgumenstOfIntersection();
			if (!SynchronizedFilter(objectPair, timeStep) || ProximityFilter(objectPair))
				objectPair.collision = true;
		}

		return (!objectPair.collision);
	}
};

__device__ double CollisionRate(CollisionPair &objectPair, double pAThreshold)
{
	double collisionRate, boundingRadii, relativeVelocity;
	vector3D velocityI, velocityJ;
	
	velocityI = objectPair.primary.GetVelocity();
	velocityJ = objectPair.secondary.GetVelocity();

	relativeVelocity = velocityI.CalculateRelativeVector(velocityJ).vectorNorm();
	boundingRadii = max(pAThreshold, objectPair.GetBoundingRadii());
	objectPair.SetRelativeVelocity(relativeVelocity);
	//sinAngle = velocityI.VectorCrossProduct(velocityJ).vectorNorm() / (velocityI.vectorNorm() * velocityJ.vectorNorm());

	// OT collision rate
	if (boundingRadii > objectPair.minSeperation)
		collisionRate = Pi * boundingRadii * relativeVelocity /
		(2 * velocityI.VectorCrossProduct(velocityJ).vectorNorm()  * objectPair.primary.GetPeriod() * objectPair.secondary.GetPeriod());
	else
		collisionRate = 0;

	return collisionRate;
}
struct MinSeperation {
	__device__ void operator()(CollisionPair& objectPair) {
		objectPair.minSeperation = objectPair.CalculateMinimumSeparation();

		objectPair.altitude = objectPair.primary.GetElements().GetRadialPosition();
	}
};

struct CollisionRateKernel {
		double timeStep, pAThreshold;
		CollisionRateKernel(double timestep, double threshold) {
			timeStep = timestep;
			pAThreshold = threshold;
		}
		__device__ void operator()(CollisionPair& objectPair) {
			objectPair.probability = timeStep * CollisionRate(objectPair, pAThreshold);
			
		}
};

__host__ void OrbitTrace::MainCollision_GPU(DebrisPopulation & population, double timestep)
{
	double mass, tempProbability, epoch = population.GetEpoch();
	Event tempEvent;
	thrust::device_vector<CollisionPair> pairList;

	// Filter Cube List
	pairList = CreatePairList_GPU(population);
	timeStep = timestep;
	//unsigned int numThreads, numBlocks;
	//computeGridSize(pairList.size(), 256, numBlocks, numThreads);
	//TODO - Add code for GPU use
	
	thrust::for_each(thrust::device, pairList.begin(), pairList.begin(), CollisionFilterKernel(timestep));
	
	dvit collisionEnd = thrust::remove_if(thrust::device, pairList.begin(), pairList.end(), NotCollision());
	pairList.erase(collisionEnd, pairList.end());

	thrust::for_each(thrust::device, pairList.begin(), pairList.end(), MinSeperation());
	thrust::for_each(thrust::device, pairList.begin(), pairList.end(), CollisionRateKernel(timestep, pAThreshold));

	//thrust::host_vector<CollisionPair> pairList_local = pairList;

	for (int i = 0; i < pairList.size(); i++) {
		CollisionPair objectPair = pairList[i];
		tempProbability = objectPair.probability;

		mass = objectPair.primary.GetMass() + objectPair.secondary.GetMass();
		tempEvent = Event(epoch, objectPair.primaryID, objectPair.secondaryID, objectPair.GetRelativeVelocity(), mass, objectPair.altitude);
		//	-- Determine if collision occurs through MC (random number generation)
		if (outputProbabilities && tempProbability > 0)
		{
			//	-- Store collision probability
			newCollisionProbabilities.push_back(tempProbability);
			newCollisionList.push_back(tempEvent);

		}
		else
		{
			if (DetermineCollision(tempProbability))
			{
				// Store Collisions 
				newCollisionList.push_back(tempEvent);
			}
		}
	}

	elapsedTime += timeStep;

}

