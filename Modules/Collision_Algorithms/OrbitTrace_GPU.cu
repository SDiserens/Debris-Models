#include "hip/hip_runtime.h"
// OrbitTrace.cpp : contains the implementation of the GPU elements of the Orbit Trace collision algorithm.
//

#include "stdafx.h"
#include "OrbitTrace.h"

// CUDA standard includes
#include <windows.h>
#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>
#include ""
#include <thrust\for_each.h>
#include <thrust\execution_policy.h>
#include <thrust\device_vector.h>

//Round a / b to nearest higher integer value
unsigned int  iDivUp(unsigned int  a, unsigned int  b)
{
	return (a % b != 0) ? (a / b + 1) : (a / b);
}

// compute grid and thread block size for a given number of elements
void computeGridSize(unsigned int n, unsigned int blockSize, unsigned int  &numBlocks, unsigned int  &numThreads)
{
	numThreads = min(blockSize, n);
	numBlocks = iDivUp(n, numThreads);
}

__device__ double CollisionRate(CollisionPair &objectPair, int MOIDtype, double pAThreshold)
{
	double collisionRate, boundingRadii, minSeperation, relativeVelocity;
	vector3D velocityI, velocityJ;

	switch (MOIDtype) {
	case 0: minSeperation = objectPair.CalculateMinimumSeparation();
	case 1: minSeperation = objectPair.CalculateMinimumSeparation_DL();
	case 2: minSeperation = objectPair.CalculateMinimumSeparation_MOID();
	}


	velocityI = objectPair.primary.GetVelocity();
	velocityJ = objectPair.secondary.GetVelocity();

	relativeVelocity = velocityI.CalculateRelativeVector(velocityJ).vectorNorm();
	boundingRadii = max(pAThreshold, objectPair.GetBoundingRadii());
	objectPair.SetRelativeVelocity(relativeVelocity);
	//sinAngle = velocityI.VectorCrossProduct(velocityJ).vectorNorm() / (velocityI.vectorNorm() * velocityJ.vectorNorm());

	// OT collision rate
	if (boundingRadii > minSeperation)
		collisionRate = Pi * boundingRadii * relativeVelocity /
		(2 * velocityI.VectorCrossProduct(velocityJ).vectorNorm()  * objectPair.primary.GetPeriod() * objectPair.secondary.GetPeriod());
	else
		collisionRate = 0;

	return collisionRate;
}
__host__ thrust::device_vector<CollisionPair> OrbitTrace::CreatePairList_GPU(DebrisPopulation & population)
{
	thrust::device_vector<CollisionPair> pairList;
	// TODO - GPU code for creating pairList
	//Talk to Pete about i, j where i < j < N
	for (auto it = population.population.begin(); it != population.population.end(); it++)
	{
		// For each subsequent object
		auto jt = it;
		for (++jt; jt != population.population.end(); ++jt)
		{
			/// Add pair to list
			//DebrisObject& primaryObject(population.Ge), secondaryObject;
			CollisionPair pair(it->second, jt->second);
			if (PerigeeApogeeTest(pair))
				pairList.push_back(pair);
			else
				pair.~CollisionPair();
		}
	}

	return pairList;
}

/*
//TODO - add device function  to operate on each collision pair
__device__ void OrbitTraceAlgorithm(list<CollisionPair>& pairList) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	
}

__host__ void OrbitTrace::MainCollision_GPU_Cuda(DebrisPopulation & population, double timestep)
{
	double tempProbability, collisionRate, altitude, mass;
	list<CollisionPair> pairList;
	pair<long, long> pairID;
	bool collision;

	// Filter Cube List
	pairList = CreatePairList_GPU(population);
	timeStep = timestep;
	unsigned int numThreads, numBlocks;
	computeGridSize(pairList.size(), 256, numBlocks, numThreads);
	//TODO - Add code for GPU use
		// 1D iteration over pairList
		//int index = blockIdx.x * blockDim.x + threadIdx.x;

}
*/
__device__ bool HeadOnFilter(CollisionPair objectPair)
{
	bool headOn = false;
	double deltaW;
	double eLimitP = objectPair.GetBoundingRadii() / objectPair.primary.GetElements().semiMajorAxis;
	double eLimitS = objectPair.GetBoundingRadii() / objectPair.secondary.GetElements().semiMajorAxis;
	// OT Head on filter
	if ((objectPair.primary.GetElements().eccentricity <= eLimitP) && (objectPair.secondary.GetElements().eccentricity <= eLimitS))
		headOn = true;
	else
	{
		deltaW = abs(Pi - objectPair.primary.GetElements().argPerigee - objectPair.secondary.GetElements().argPerigee);
		if (deltaW <= 1)
			headOn = true;
		else if (Tau - deltaW <= 1)
			headOn = true;
	}

	return headOn;
}

__device__ bool SynchronizedFilter(CollisionPair objectPair, double timeStep)
{
	double meanMotionP, meanMotionS, driftAngle;
	// OT synch filter
	meanMotionP = Tau / objectPair.primary.GetPeriod();
	meanMotionS = Tau / objectPair.secondary.GetPeriod();

	driftAngle = abs(meanMotionP - meanMotionS) * timeStep;
	return (driftAngle >= Tau);
}

__device__ bool ProximityFilter(CollisionPair objectPair)
{
	//  OT  proximity filter
	double deltaMP, deltaMS, deltaMAngle, deltaMLinear, combinedSemiMajorAxis;
	OrbitalAnomalies anomaliesP, anomaliesS;

	anomaliesP.SetTrueAnomaly(objectPair.approachAnomalyP);
	anomaliesS.SetTrueAnomaly(objectPair.approachAnomalyS);

	deltaMP = abs(anomaliesP.GetMeanAnomaly(objectPair.primary.GetElements().eccentricity) - objectPair.primary.GetElements().GetMeanAnomaly());
	deltaMS = abs(anomaliesS.GetMeanAnomaly(objectPair.secondary.GetElements().eccentricity) - objectPair.secondary.GetElements().GetMeanAnomaly());

	combinedSemiMajorAxis = (objectPair.primary.GetElements().semiMajorAxis + objectPair.secondary.GetElements().semiMajorAxis) / 2;
	deltaMAngle = abs(deltaMP - deltaMS);
	deltaMLinear = deltaMAngle * combinedSemiMajorAxis;

	return (deltaMLinear <= objectPair.GetBoundingRadii());
}



struct CollisionSteps {
	double timeStep, pAThreshold;
	int MOIDtype;
	CollisionSteps(double timestep, int moid, double threshold) {
		timeStep = timestep;
		MOIDtype = moid;
		pAThreshold = threshold;
	}
__device__ void operator()(CollisionPair& objectPair) {
	objectPair.collision = false;

	objectPair.CalculateRelativeInclination();
	double combinedSemiMajorAxis = objectPair.primary.GetElements().semiMajorAxis + objectPair.secondary.GetElements().semiMajorAxis;
	bool coplanar = objectPair.GetRelativeInclination() <= (2 * asin(objectPair.GetBoundingRadii() / combinedSemiMajorAxis));
	objectPair.coplanar = coplanar;

	if (coplanar)
	{
		// Calculate orbit intersections for coplanar
		objectPair.CalculateArgumenstOfIntersectionCoplanar();
		if (HeadOnFilter(objectPair) || !SynchronizedFilter(objectPair, timeStep) || ProximityFilter(objectPair))
			objectPair.collision = true;
	}
	else
	{
		// Calculate intersections for non coplanar
		objectPair.CalculateArgumenstOfIntersection();
		if (!SynchronizedFilter(objectPair, timeStep) || ProximityFilter(objectPair))
			objectPair.collision = true;
	}

	if (objectPair.collision)
	{
		double  altitude, mass;
		thrust::pair<long, long> pairID;
		objectPair.probability = timeStep * CollisionRate(objectPair, MOIDtype, pAThreshold);
		pairID = thrust::make_pair(objectPair.primaryID, objectPair.secondaryID);

		altitude = objectPair.primary.GetElements().GetRadialPosition();
		mass = objectPair.primary.GetMass() + objectPair.secondary.GetMass();
		objectPair.tempEvent = Event(0, pairID.first, pairID.second, objectPair.GetRelativeVelocity(), mass, altitude);
	}
	else
		objectPair.probability = 0;
}
};

__host__ void OrbitTrace::MainCollision_GPU(DebrisPopulation & population, double timestep)
{
	double tempProbability, epoch = population.GetEpoch();
	thrust::device_vector<CollisionPair> pairList;

	// Filter Cube List
	pairList = CreatePairList_GPU(population);
	timeStep = timestep;
	unsigned int numThreads, numBlocks;
	computeGridSize(pairList.size(), 256, numBlocks, numThreads);
	//TODO - Add code for GPU use
	thrust::for_each(thrust::device, pairList.begin(), pairList.end(), CollisionSteps(timestep, MOIDtype, pAThreshold));
	
		
	for (int i = 0; i < pairList.size(); i++) {
		CollisionPair objectPair = pairList[i];
		tempProbability = objectPair.probability;
		//	-- Determine if collision occurs through MC (random number generation)
		if (outputProbabilities && tempProbability > 0)
		{
			//	-- Store collision probability
			objectPair.tempEvent.SetEpoch(epoch);
			newCollisionProbabilities.push_back(tempProbability);
			newCollisionList.push_back(objectPair.tempEvent);

		}
		else
		{
			if (DetermineCollision(tempProbability))
			{
				// Store Collisions 
				objectPair.tempEvent.SetEpoch(epoch);
				newCollisionList.push_back(objectPair.tempEvent);
			}
		}
	}

	elapsedTime += timeStep;

}

