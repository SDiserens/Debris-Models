#include "hip/hip_runtime.h"
#include "stdafx.h"
#include "CollisionPair.cuh"

CollisionPair::CollisionPair()
{
}

CollisionPair::CollisionPair(DebrisObject& objectI, DebrisObject& objectJ)
{
	primaryElements = objectI.GetElements();
	secondaryElements = objectJ.GetElements();
	primaryID = objectI.GetID();
	secondaryID = objectJ.GetID();
	primaryAnomaly = objectI.GetEpochAnomaly();
	secondaryAnomaly = objectJ.GetEpochAnomaly();
	approachAnomalyP = primaryElements.GetTrueAnomaly();
	approachAnomalyS = secondaryElements.GetTrueAnomaly();
	primaryMass = objectI.GetMass();
	secondaryMass = objectJ.GetMass();
	//CalculateRelativeInclination();
	//CalculateArgumenstOfIntersection();
	boundingRadii = (objectI.GetRadius() + objectJ.GetRadius()) * 0.001;
	overlapCount = 1;

	CalculateRelativeInclination();
}

CollisionPair::CollisionPair(long IDI, long IDJ)
{
	primaryID = IDI;
	secondaryID = IDJ;
}

double CollisionPair::GetRelativeInclination()
{
	return relativeInclination;
}

CUDA_CALLABLE_MEMBER void CollisionPair::SetCollisionPair(DebrisObject objectI, DebrisObject objectJ)
{

	primaryElements = objectI.GetElements();
	secondaryElements = objectJ.GetElements();
	primaryID = objectI.GetID();
	secondaryID = objectJ.GetID();
	primaryAnomaly = objectI.GetEpochAnomaly();
	secondaryAnomaly = objectJ.GetEpochAnomaly();
	primaryMass = objectI.GetMass();
	secondaryMass = objectJ.GetMass();
	//CalculateRelativeInclination();
	//CalculateArgumenstOfIntersection();
	boundingRadii = (objectI.GetRadius() + objectJ.GetRadius()) * 0.001;
	overlapCount = 1;

	CalculateRelativeInclination();
}

vector<double> CollisionPair::CalculateAngularWindowPrimary(double distance)
{
	return CalculateAngularWindow(primaryElements, distance, deltaPrimary);
}

vector<double> CollisionPair::CalculateAngularWindowSecondary(double distance)
{
	return CalculateAngularWindow(secondaryElements, distance, deltaSecondary);
}

vector3D CollisionPair::GetPrimaryPositionAtTime(double timeFromEpoch)
{
	// position at time
	double meanAnomaly = TauRange(primaryAnomaly + Tau * timeFromEpoch / primaryElements.CalculatePeriod());
	primaryElements.SetMeanAnomaly(meanAnomaly);
	return primaryElements.GetPosition();
}

vector3D CollisionPair::GetPrimaryVelocityAtTime(double timeFromEpoch)
{
	// velcoity at time
	double meanAnomaly = TauRange(primaryAnomaly + Tau * timeFromEpoch / primaryElements.CalculatePeriod());
	primaryElements.SetMeanAnomaly(meanAnomaly);
	return primaryElements.GetVelocity();
}

vector3D CollisionPair::GetSecondaryPositionAtTime(double timeFromEpoch)
{
	// position at time
	double meanAnomaly = TauRange(secondaryAnomaly + Tau * timeFromEpoch / secondaryElements.CalculatePeriod());
	secondaryElements.SetMeanAnomaly(meanAnomaly);
	return secondaryElements.GetPosition();
}

vector3D CollisionPair::GetSecondaryVelocityAtTime(double timeFromEpoch)
{
	// velcoity at time
	double meanAnomaly = TauRange(secondaryAnomaly + Tau * timeFromEpoch / secondaryElements.CalculatePeriod());
	secondaryElements.SetMeanAnomaly(meanAnomaly);
	return secondaryElements.GetVelocity();
}

void CollisionPair::GenerateArgumenstOfIntersection()
{
	CalculateRelativeInclination();
	coplanar = (relativeInclination <= (2 * asin(boundingRadii / (primaryElements.semiMajorAxis + secondaryElements.semiMajorAxis))));
	if (coplanar)
		CalculateArgumenstOfIntersectionCoplanar();
	else
		CalculateArgumenstOfIntersection();
}

double CollisionPair::CalculateMinimumSeparation_MOID()
{
	moid_data_t mdata;

 	double distance = find_moid_full(primaryElements, secondaryElements, &mdata);

	approachAnomalyP = mdata.obj1_true_anom;
	approachAnomalyS = mdata.obj2_true_anom;
	primaryElements.SetTrueAnomaly(mdata.obj1_true_anom);
	secondaryElements.SetTrueAnomaly(approachAnomalyS);

	SetCollisionAltitude(primaryElements.GetRadialPosition());

	return distance;
}

double CollisionPair::CalculateMinimumSeparation_DL()
{
	COrbitData<double> object1(primaryElements.semiMajorAxis, primaryElements.eccentricity, primaryElements.inclination, primaryElements.rightAscension, primaryElements.argPerigee);
	COrbitData<double> object2(secondaryElements.semiMajorAxis, secondaryElements.eccentricity, secondaryElements.inclination, secondaryElements.rightAscension, secondaryElements.argPerigee);

	double max_root_error, min_root_error, max_anom_error;
	detect_suitable_options(max_root_error, min_root_error, max_anom_error);
	SMOIDResult<double> result = MOID_fast(object1, object2, max_root_error, min_root_error);

	if (!result.good) {
		result = MOID_fast(object2, object1, max_root_error, min_root_error);

		if (!result.good) {
			unsigned int densities[4] = { 1000, 30, 3, 0 };
			result = MOID_direct_search(object1, object2, densities, 0.01, max_anom_error);
		}
	}
	primaryElements.SetEccentricAnomaly(result.u1);
	secondaryElements.SetEccentricAnomaly(result.u2);

	approachAnomalyP = primaryElements.GetTrueAnomaly();
	approachAnomalyS = secondaryElements.GetTrueAnomaly();
	SetCollisionAltitude(primaryElements.GetRadialPosition());

	return result.distance;
}


double CollisionPair::CalculateMinimumSeparation()
{
	double trueAnomalyP, trueAnomalyS, seperation,  eP, eS;

	trueAnomalyP = TauRange(deltaPrimary - primaryElements.argPerigee);
	trueAnomalyS = TauRange(deltaSecondary - secondaryElements.argPerigee);
	eP = primaryElements.eccentricity;
	eS = secondaryElements.eccentricity;
	primaryElements.SetTrueAnomaly(trueAnomalyP);
	secondaryElements.SetTrueAnomaly(trueAnomalyS);

	seperation = primaryElements.GetPosition().CalculateRelativeVector(secondaryElements.GetPosition()).vectorNorm();

	// Find closest approach for elliptical orbits
	if (eP != 0 || eS != 0)
	{
		int error1, error2;
		double seperation1, seperation2, baseSeperation2, altTrueAnomalyS1, altTrueAnomalyP1, altTrueAnomalyS2, altTrueAnomalyP2;
		auto NewtonSeperation = [&](double &trueP, double &trueS)
		{

			int it = 0;
			double F, G, FdfP, FdfS, GdfP, GdfS;
			double uRP, uRS, A, B, C, D, axP, ayP, axS, ayS;
			double rP, rS, sinURP, sinURS, cosURP, cosURS, EP, ES, sinVP, cosVP, sinVS, cosVS;
			double tempAnomalyP, tempAnomalyS, circularAnomalyP, circularAnomalyS, cosRI;
			double k = 2.0, h = 2.0;
			double base, baseMin = 999;

			circularAnomalyP = tempAnomalyP = trueP;
			circularAnomalyS = tempAnomalyS = trueS;
			cosRI = cos(relativeInclination);

			axP = eP * cos(-circularAnomalyP);
			ayP = sqrt(eP * eP - axP * axP); //eP * sin(-circularAnomalyP);
			axS = eS * cos(-circularAnomalyS);
			ayS = sqrt(eS * eS - axS * axS); // eS * sin(-circularAnomalyS);

											 // Min Sep newton method
			while ((abs(h) >= NEWTONTOLERANCE || abs(k) >= NEWTONTOLERANCE) && (it < NEWTONMAXITERATIONS))
			{
				rP = primaryElements.GetRadialPosition(tempAnomalyP);
				rS = secondaryElements.GetRadialPosition(tempAnomalyS);
				uRP = tempAnomalyP - circularAnomalyP;
				uRS = tempAnomalyS - circularAnomalyS;

				sinURP = sin(uRP);
				cosURP = sqrt(1 - sinURP * sinURP);  //cos(uRP);
				sinURS = sin(uRS);
				cosURS = sqrt(1 - sinURS * sinURS);  //cos(uRS);
				sinVP = sin(tempAnomalyP);
				cosVP = sqrt(1 - sinVP * sinVP); // cos(tempAnomalyP);
				sinVS = sin(tempAnomalyS);
				cosVS = sqrt(1 - sinVS * sinVS); // cos(tempAnomalyS);

				A = sinURP + ayP;
				C = sinURS + ayS;
				B = cosURP + axP;
				D = cosURS + axS;

				EP = atan2(sqrt(1 - eP * eP) * sinVP, eP + cosVP);
				ES = atan2(sqrt(1 - eS * eS) * sinVS, eS + cosVS);

				F = rP * eP * sinVP + rS * (A * cosURS - B * cosRI * sinURS);
				G = rS * eS * sinVS + rP * (C * cosURP - D * cosRI * sinURP);

				FdfP = rP * eP * cos(EP) + rS * (cosURP * cosURS + sinURP * sinURS * cosRI);
				FdfS = -rS / (1 + eS * cosVS) * (A * C + B * D * cosRI);
				GdfP = -rP / (1 + eP * cosVP) * (A * C + B * D * cosRI);
				GdfS = rS * eS * cos(ES) + rP * (cosURP * cosURS + sinURP * sinURS * cosRI);

				base = (FdfS*GdfP - FdfP*GdfS);
				if (abs(base) < baseMin)
					baseMin = abs(base);

				h = (F * GdfS - G * FdfS) / base;
				k = (G * FdfP - F * GdfP) / base;


				if (it > 1 && abs(base) < 25)
				{
					//Implement line search
					it = NEWTONMAXITERATIONS;
					break;
				}

				// Update values
				tempAnomalyP = TauRange(tempAnomalyP + h);
				tempAnomalyS = TauRange(tempAnomalyS + k);
				++it;
			}
			//TODO Handle case where iterations reached
			if (it == NEWTONMAXITERATIONS + 1)
				it = it;
			else
			{
				trueP = tempAnomalyP;
				trueS = tempAnomalyS;
			}
			if (it == NEWTONMAXITERATIONS)
				return 1;
			else
				return 0;
		};
		
		altTrueAnomalyP1 = trueAnomalyP;
		altTrueAnomalyS1 = trueAnomalyS;
		error1 = NewtonSeperation(altTrueAnomalyS1, altTrueAnomalyS2);
		primaryElements.SetTrueAnomaly(altTrueAnomalyS1);
		secondaryElements.SetTrueAnomaly(altTrueAnomalyS2);
		
		seperation1 = primaryElements.GetPosition().CalculateRelativeVector(secondaryElements.GetPosition()).vectorNorm();
		if (!error1 && seperation1 < seperation)
		{
			seperation = seperation1;
			trueAnomalyP = altTrueAnomalyP1;
			trueAnomalyS = altTrueAnomalyS2;
		}

		// Test second intersection point

		if (error1 || coplanar)
		{
		altTrueAnomalyP2 = TauRange(deltaPrimary2 - primaryElements.argPerigee);
		altTrueAnomalyS2 = TauRange(deltaSecondary2 - secondaryElements.argPerigee);
		}
		else
		{
		altTrueAnomalyP2 = TauRange(trueAnomalyP + Pi);
		altTrueAnomalyS2 = TauRange(trueAnomalyS + Pi);
		}

		primaryElements.SetTrueAnomaly(altTrueAnomalyP2);
		secondaryElements.SetTrueAnomaly(altTrueAnomalyS2);
		baseSeperation2 = primaryElements.GetPosition().CalculateRelativeVector(secondaryElements.GetPosition()).vectorNorm();
		if (baseSeperation2 < seperation)
		{
			seperation = baseSeperation2;
			trueAnomalyP = altTrueAnomalyP2;
			trueAnomalyS = altTrueAnomalyS2;
		}

		error2 = NewtonSeperation(altTrueAnomalyP2, altTrueAnomalyS2);
		primaryElements.SetTrueAnomaly(altTrueAnomalyP2);
		secondaryElements.SetTrueAnomaly(altTrueAnomalyS2);
		seperation2 = primaryElements.GetPosition().CalculateRelativeVector(secondaryElements.GetPosition()).vectorNorm();

		if (!error2 && seperation2 < seperation)
		{
			seperation = seperation2;
			trueAnomalyP = altTrueAnomalyP2;
			trueAnomalyS = altTrueAnomalyS2;
		}
		else {
			primaryElements.SetTrueAnomaly(trueAnomalyP);
			secondaryElements.SetTrueAnomaly(trueAnomalyS);
		}
	}

	approachAnomalyP = trueAnomalyP;
	approachAnomalyS = trueAnomalyS;
	SetCollisionAltitude(primaryElements.GetRadialPosition());
	return seperation;
}


double CollisionPair::GetBoundingRadii()
{
	return boundingRadii; // Combined radii in kilometres;
}

double CollisionPair::GetCollisionAltitude()
{
	return collisionAltitude;
}

void CollisionPair::SetCollisionAltitude(double altitude)
{
	collisionAltitude = altitude;
}

void CollisionPair::SetRelativeVelocity(double relV)
{
	relativeVelocity = relV;
}

double CollisionPair::GetRelativeVelocity()
{
	return relativeVelocity;
}

double CollisionPair::CalculateSeparationAtTime(double timeFromEpoch)
{
	double seperation;
	vector3D positionP = GetPrimaryPositionAtTime(timeFromEpoch);
	vector3D positionS = GetSecondaryPositionAtTime(timeFromEpoch);

	//closest approach distance
	seperation = positionP.CalculateRelativeVector(positionS).vectorNorm();
	collisionAltitude = positionP.vectorNorm();
	return seperation;
}

double CollisionPair::GetMinSeparation()
{
	double seperation;
	primaryElements.SetTrueAnomaly(approachAnomalyP);
	secondaryElements.SetTrueAnomaly(approachAnomalyS);
	vector3D positionP = primaryElements.GetPosition();
	vector3D positionS = secondaryElements.GetPosition();

	//closest approach distance
	seperation = positionP.CalculateRelativeVector(positionS).vectorNorm();
	return seperation;
}

void CollisionPair::CalculateRelativeInclination()
{
	// Calculate relative inclination
	/*
	sin IR = |cross(hP, hC)|

	where r hP is the normal to the orbit plane of the primary object
	*/
	vector3D hP = primaryElements.GetNormalVector();
	vector3D hS = secondaryElements.GetNormalVector();
	double k = hP.VectorCrossProduct(hS).vectorNorm();

	relativeInclination = asin(k);
}


void CollisionPair::CalculateArgumenstOfIntersection()
{
	// Arguments of intersection
	double cscIr, sinIp, sinIs, cosIp, cosIs, sinOmDif, cosOmDif, XP, XS, YP, YS;

	cscIr = 1 / sin(relativeInclination);
	sinIp = sin(primaryElements.inclination);
	cosIp = cos(primaryElements.inclination);
	sinIs = sin(secondaryElements.inclination);
	cosIs = cos(secondaryElements.inclination);
	sinOmDif = sin(primaryElements.rightAscension - secondaryElements.rightAscension);
	cosOmDif = cos(primaryElements.rightAscension - secondaryElements.rightAscension);

	XP = cscIr*(sinIp*cosIs - sinIs*cosIp*cosOmDif);
	XS = cscIr*(sinIp*cosIs*cosOmDif - sinIs*cosIp);
	YP = cscIr * sinIs * sinOmDif;
	YS = cscIr * sinIp * sinOmDif;

	deltaPrimary = asin(YP);
	if (XP < 0)
		deltaPrimary = Pi - deltaPrimary;
	else if (YP < 0)
		deltaPrimary += Tau;

	deltaSecondary = asin(YS);
	if (XS < 0)
		deltaSecondary = Pi - deltaSecondary;
	else if (YS < 0)
		deltaSecondary += Tau;

	deltaPrimary2 =  TauRange(deltaPrimary + Pi);
	deltaSecondary2 = TauRange(deltaSecondary + Pi);

	approachAnomalyP = TauRange(deltaPrimary - primaryElements.argPerigee);
	approachAnomalyS = TauRange(deltaSecondary - secondaryElements.argPerigee);
}

void CollisionPair::CalculateArgumenstOfIntersectionCoplanar()
{
	double cP, cS, A, B, C, X, X2, Yplus, Yminus;

	cP = primaryElements.semiMajorAxis * (1 - primaryElements.eccentricity * primaryElements.eccentricity);
	cS = secondaryElements.semiMajorAxis * (1 - secondaryElements.eccentricity * secondaryElements.eccentricity);

	A = cP - cS;
	B = cP * secondaryElements.eccentricity * cos(secondaryElements.argPerigee) - cS * primaryElements.eccentricity * cos(primaryElements.argPerigee);
	C = cP * secondaryElements.eccentricity * sin(secondaryElements.argPerigee) - cS * primaryElements.eccentricity * sin(primaryElements.argPerigee);

	Yplus = (C + sqrt(C*C + B*B - A*A)) / (A - B);

	Yminus = (C - sqrt(C*C + B*B - A*A)) / (A - B);

	X = 2 * atan(Yplus);
	X2 = 2 * atan(Yminus);

	// (Rate of change of seperations?)
	deltaPrimary = deltaSecondary = TauRange(X);
	deltaPrimary2 = deltaSecondary2 = TauRange(X2);

	approachAnomalyP = TauRange(deltaPrimary - primaryElements.argPerigee);
	approachAnomalyS = TauRange(deltaSecondary - secondaryElements.argPerigee);
}

vector<double> CollisionPair::CalculateAngularWindow(OrbitalElements & elements, double distance, double delta)
{
	vector<double> angleWindows;
	double circularAnomaly, alpha, aX, aY, Q, Qroot, cosUrMinus, cosUrPlus, windowStart, windowEnd, windowStart2, windowEnd2;

	// Calculate Angular Windows
	circularAnomaly = delta - elements.argPerigee;
	alpha = elements.semiMajorAxis * (1 - elements.eccentricity * elements.eccentricity) * sin(relativeInclination);
	aX = elements.eccentricity * cos(-circularAnomaly);
	aY = elements.eccentricity * sin(-circularAnomaly);
	Q = alpha * (alpha - 2 * distance * aY) - (1 - elements.eccentricity * elements.eccentricity) * distance * distance;


	if (Q < 0)
	{
		// Handle  coplanar case
		angleWindows.push_back(-1.0);
		return angleWindows;
	}
	else if (Q == 0)
	{
		// Check for singular case where close approach at perige
		Qroot = 0;
		cosUrPlus = -aX;
		cosUrMinus = 0;
	}
	else
	{
		Qroot = sqrt(Q);
		cosUrMinus = (-distance * distance * aX - (alpha - distance * aY) * Qroot) / (Q + distance * distance);
		cosUrPlus = (-distance * distance * aX + (alpha - distance * aY) * Qroot) / (Q + distance * distance);
	}

	// Handle  coplanar case
	if (abs(cosUrMinus) > 1)
	{
		angleWindows.push_back(-2.0);
		return angleWindows;
	}
	else if (abs(cosUrPlus) > 1)
	{
		angleWindows.push_back(-3.0);
		return angleWindows;
	}

	windowEnd = acos(cosUrPlus);
	windowStart = 0 - windowEnd;

	windowStart -= circularAnomaly;
	windowEnd -= circularAnomaly;

	if (windowEnd < 0)
	{
		windowStart += Tau;
		windowEnd += Tau;
	}

	angleWindows.insert(angleWindows.end(), { windowStart, windowEnd });
	if (Q != 0)
	{
		windowStart2 = acos(cosUrMinus);
		windowEnd2 = Tau - windowStart2;
		windowStart2 -= circularAnomaly;
		windowEnd2 -= circularAnomaly;

		if (windowEnd2 < 0)
		{
			windowStart2 += Tau;
			windowEnd2 += Tau;
		}

		angleWindows.insert(angleWindows.end(), { windowStart2, windowEnd2 });
	}

	return angleWindows;
}
